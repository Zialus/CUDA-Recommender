#include "hip/hip_runtime.h"
#include "CCD_CUDA.h"

__global__ void RankOneUpdate_v_kernel(const long Rcols,
                                       const long* Rcol_ptr,
                                       const long* Rrow_idx,
                                       const float* Rval,

                                       float* u,
                                       float* v,
                                       const float lambda,
                                       const int do_nmf
) {
    long thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    long total_threads = blockDim.x * gridDim.x;

    for (long c = thread_id; c < Rcols; c += total_threads) {
        v[c] = RankOneUpdate_dev(Rcol_ptr, Rrow_idx, Rval, c, u,
                                 lambda * (Rcol_ptr[c + 1] - Rcol_ptr[c]), do_nmf);
    }

}

__global__ void RankOneUpdate_u_kernel(const long Rcols_t,
                                       const long* Rcol_ptr_t,
                                       const long* Rrow_idx_t,
                                       const float* Rval_t,

                                       float* u,
                                       float* v,
                                       const float lambda,
                                       const int do_nmf
) {
    long thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    long total_threads = blockDim.x * gridDim.x;

    for (long c = thread_id; c < Rcols_t; c += total_threads) {
        u[c] = RankOneUpdate_dev(Rcol_ptr_t, Rrow_idx_t, Rval_t, c, v,
                                 lambda * (Rcol_ptr_t[c + 1] - Rcol_ptr_t[c]), do_nmf);
    }

}

__device__ float RankOneUpdate_dev(const long* Rcol_ptr,
                                   const long* Rrow_idx,
                                   const float* Rval,

                                   const long j,
                                   const float* u_vec_t,

                                   const float lambda,
                                   const int do_nmf
) {
    float g = 0, h = lambda;
    if (Rcol_ptr[j + 1] == Rcol_ptr[j]) { return 0; }

    for (long idx = Rcol_ptr[j]; idx < Rcol_ptr[j + 1]; ++idx) {
        long i = Rrow_idx[idx];
        g += u_vec_t[i] * Rval[idx];
        h += u_vec_t[i] * u_vec_t[i];
    }

    float newvj = g / h;
    if (do_nmf > 0 & newvj < 0) {
        newvj = 0;
    }
    return newvj;
}

__global__ void UpdateRating_DUAL_kernel_NoLoss(const long Rcols,
                                                const long* Rcol_ptr,
                                                const long* Rrow_idx,
                                                float* Rval,
                                                const float* Wt_vec_t,
                                                const float* Ht_vec_t,
                                                const bool add,

                                                const long Rcols_t,
                                                const long* Rcol_ptr_t,
                                                const long* Rrow_idx_t,
                                                float* Rval_t,
                                                const bool add_t
) {
    long thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    long total_threads = blockDim.x * gridDim.x;

    for (long i = thread_id; i < Rcols; i += total_threads) {
        if (add) {
            float Htc = Ht_vec_t[i];
            for (long idx = Rcol_ptr[i]; idx < Rcol_ptr[i + 1]; ++idx) {
                Rval[idx] += Wt_vec_t[Rrow_idx[idx]] * Htc; //change R.val
            }
        } else {
            float Htc = Ht_vec_t[i];
            for (long idx = Rcol_ptr[i]; idx < Rcol_ptr[i + 1]; ++idx) {
                Rval[idx] -= Wt_vec_t[Rrow_idx[idx]] * Htc; //change R.val
            }
        }
    }

    for (long i = thread_id; i < Rcols_t; i += total_threads) {
        if (add_t) {
            float Htc = Wt_vec_t[i];
            for (long idx = Rcol_ptr_t[i]; idx < Rcol_ptr_t[i + 1]; ++idx) {
                Rval_t[idx] += Ht_vec_t[Rrow_idx_t[idx]] * Htc; //change R.val
            }
        } else {
            float Htc = Wt_vec_t[i];
            for (long idx = Rcol_ptr_t[i]; idx < Rcol_ptr_t[i + 1]; ++idx) {
                Rval_t[idx] -= Ht_vec_t[Rrow_idx_t[idx]] * Htc; //change R.val
            }
        }
    }
}

__global__ void UpdateRating_W_kernel(const long Rcols,
                                      const long* Rcol_ptr,
                                      const long* Rrow_idx,
                                      float* Rval,

                                      const float* Wt_vec_t,
                                      const float* Ht_vec_t,
                                      const bool add


) {
    long thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    long total_threads = blockDim.x * gridDim.x;

    for (long i = thread_id; i < Rcols; i += total_threads) {
        if (add) {
            float Htc = Ht_vec_t[i];
            for (long idx = Rcol_ptr[i]; idx < Rcol_ptr[i + 1]; ++idx) {
                Rval[idx] += Wt_vec_t[Rrow_idx[idx]] * Htc; //change R.val
            }
        } else {
            float Htc = Ht_vec_t[i];
            for (long idx = Rcol_ptr[i]; idx < Rcol_ptr[i + 1]; ++idx) {
                Rval[idx] -= Wt_vec_t[Rrow_idx[idx]] * Htc; //change R.val
            }
        }
    }

}

__global__ void UpdateRating_H_kernel(const long Rcols_t,
                                      const long* Rcol_ptr_t,
                                      const long* Rrow_idx_t,
                                      float* Rval_t,

                                      const float* Wt_vec_t,
                                      const float* Ht_vec_t,
                                      const bool add_t

) {
    long thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    long total_threads = blockDim.x * gridDim.x;

    for (long i = thread_id; i < Rcols_t; i += total_threads) {
        if (add_t) {
            float Htc = Wt_vec_t[i];
            for (long idx = Rcol_ptr_t[i]; idx < Rcol_ptr_t[i + 1]; ++idx) {
                Rval_t[idx] += Ht_vec_t[Rrow_idx_t[idx]] * Htc; //change R.val
            }
        } else {
            float Htc = Wt_vec_t[i];
            for (long idx = Rcol_ptr_t[i]; idx < Rcol_ptr_t[i + 1]; ++idx) {
                Rval_t[idx] -= Ht_vec_t[Rrow_idx_t[idx]] * Htc; //change R.val
            }
        }
    }
}

void kernel_wrapper_ccdpp_NV(smat_t& R, testset_t& T, mat_t& W, mat_t& H, parameter& parameters) {
    hipError_t cudaStatus;
    // Reset GPU.
    cudaStatus = hipDeviceReset();
    gpuErrchk(cudaStatus);
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    gpuErrchk(cudaStatus);

    cudaStatus = ccdpp_NV(R, T, W, H, parameters);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CCD FAILED: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipDeviceReset();
    gpuErrchk(cudaStatus);
}

inline hipError_t
updateRating(unsigned int nThreadsPerBlock, unsigned int nBlocks, const smat_t& R_C, const smat_t& Rt,
             const long* dev_Rcol_ptr, const long* dev_Rrow_idx, const long* dev_Rcol_ptr_T, const long* dev_Rrow_idx_T,
             float* dev_Rval, float* dev_Rval_t, const float* dev_Wt_vec_t, const float* dev_Ht_vec_t, const bool add,
             hipError_t& cudaStatus) {

    UpdateRating_DUAL_kernel_NoLoss<<<nBlocks, nThreadsPerBlock>>>(R_C.cols, dev_Rcol_ptr, dev_Rrow_idx,
                    dev_Rval, dev_Wt_vec_t, dev_Ht_vec_t, add, Rt.cols,
                    dev_Rcol_ptr_T, dev_Rrow_idx_T, dev_Rval_t, add);

//    UpdateRating_W_kernel<<<nBlocks, nThreadsPerBlock>>>(R_C.cols, dev_Rcol_ptr, dev_Rrow_idx,
//            dev_Rval, dev_Wt_vec_t, dev_Ht_vec_t, add);
//    UpdateRating_H_kernel<<<nBlocks, nThreadsPerBlock>>>(Rt.cols, dev_Rcol_ptr_T, dev_Rrow_idx_T,
//            dev_Rval_t, dev_Wt_vec_t, dev_Ht_vec_t, add);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    gpuErrchk(cudaStatus);
    cudaStatus = hipDeviceSynchronize();
    gpuErrchk(cudaStatus);
    return cudaStatus;
}

inline hipError_t
RankOneUpdate(const parameter& parameters, unsigned int nThreadsPerBlock, unsigned int nBlocks, float lambda,
              const smat_t& R_C, const smat_t& Rt, const long* dev_Rcol_ptr, const long* dev_Rrow_idx,
              const long* dev_Rcol_ptr_T, const long* dev_Rrow_idx_T, const float* dev_Rval,
              const float* dev_Rval_t, float* dev_Wt_vec_t,float* dev_Ht_vec_t, hipError_t& cudaStatus) {

    RankOneUpdate_v_kernel<<<nBlocks, nThreadsPerBlock>>>(R_C.cols, dev_Rcol_ptr, dev_Rrow_idx,
            dev_Rval, dev_Wt_vec_t, dev_Ht_vec_t, lambda, parameters.do_nmf);
    RankOneUpdate_u_kernel<<<nBlocks, nThreadsPerBlock>>>(Rt.cols, dev_Rcol_ptr_T, dev_Rrow_idx_T,
            dev_Rval_t, dev_Wt_vec_t, dev_Ht_vec_t, lambda, parameters.do_nmf);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    gpuErrchk(cudaStatus);
    cudaStatus = hipDeviceSynchronize();
    gpuErrchk(cudaStatus);
    return cudaStatus;
}


hipError_t ccdpp_NV(smat_t& R_C, testset_t& T, mat_t& W, mat_t& H, parameter& parameters) {
    unsigned nThreadsPerBlock = parameters.nThreadsPerBlock;
    unsigned nBlocks = parameters.nBlocks;

    hipError_t cudaStatus;

    int k = parameters.k;
    float lambda = parameters.lambda;

    // Create transpose view of R
    smat_t Rt;
    Rt = R_C.transpose();

    long* dev_Rcol_ptr = nullptr;
    long* dev_Rrow_idx = nullptr;
    long* dev_Rcol_ptr_T = nullptr;
    long* dev_Rrow_idx_T = nullptr;
    float* dev_Rval = nullptr;
    float* dev_Rval_t = nullptr;

    float* dev_Wt_vec_t = nullptr; //u
    float* dev_Ht_vec_t = nullptr; //v

    float* dev_W_ = nullptr;
    float* dev_H_ = nullptr;

    size_t nbits_W_ = R_C.rows * k * sizeof(float);
    float* W_ = (float*) malloc(nbits_W_);
    size_t nbits_H_ = R_C.cols * k * sizeof(float);
    float* H_ = (float*) malloc(nbits_H_);

    int indexPosition = 0;
    for (long i = 0; i < k; ++i) {
        for (int j = 0; j < R_C.rows; ++j) {
            W_[indexPosition] = W[i][j];
            ++indexPosition;
        }
    }

//    indexPosition = 0;
//    for (long i = 0; i < k; ++i) {
//        for (int j = 0; j < R_C.cols; ++j) {
//            H_[indexPosition] = H[i][j];
//            ++indexPosition;
//        }
//    }

    cudaStatus = hipMalloc((void**) &dev_W_, nbits_W_);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_H_, nbits_H_);
    gpuErrchk(cudaStatus);

    cudaStatus = hipMemcpy(dev_W_, W_, nbits_W_, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemset(dev_H_, 0, nbits_H_);
//    cudaStatus = hipMemcpy(dev_H_, H_, nbits_H_, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);


    cudaStatus = hipMalloc((void**) &dev_Rcol_ptr, R_C.nbits_col_ptr);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_Rrow_idx, R_C.nbits_row_idx);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_Rcol_ptr_T, Rt.nbits_col_ptr);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_Rrow_idx_T, Rt.nbits_row_idx);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_Rval, R_C.nbits_val);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_Rval_t, Rt.nbits_val);
    gpuErrchk(cudaStatus);

    cudaStatus = hipMemcpy(dev_Rcol_ptr, R_C.col_ptr, R_C.nbits_col_ptr, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_Rrow_idx, R_C.row_idx, R_C.nbits_row_idx, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_Rcol_ptr_T, Rt.col_ptr, Rt.nbits_col_ptr, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_Rrow_idx_T, Rt.row_idx, Rt.nbits_row_idx, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_Rval, R_C.val, R_C.nbits_val, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_Rval_t, Rt.val, Rt.nbits_val, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);

    float* rmse = (float*) malloc((T.nnz) * sizeof(float));

    long* d_test_row;
    long* d_test_col;
    float* d_test_val;
    float* d_pred_v;
    float* d_rmse;

    gpuErrchk(hipMalloc((void**) &d_test_row, (T.nnz + 1) * sizeof(long)));
    gpuErrchk(hipMalloc((void**) &d_test_col, (T.nnz + 1) * sizeof(long)));
    gpuErrchk(hipMalloc((void**) &d_test_val, (T.nnz + 1) * sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_pred_v, (T.nnz + 1) * sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_rmse, (T.nnz + 1) * sizeof(float)));

    gpuErrchk(hipMemcpy(d_test_row, T.test_row, (T.nnz + 1) * sizeof(long), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_test_col, T.test_col, (T.nnz + 1) * sizeof(long), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_test_val, T.test_val, (T.nnz + 1) * sizeof(float), hipMemcpyHostToDevice));

    float update_time_acc = 0;
    float rank_time_acc = 0;

    for (int oiter = 1; oiter <= parameters.maxiter; ++oiter) {

        float update_time = 0;
        float rank_time = 0;
        GpuTimer update_timer;
        GpuTimer rmse_timer;
        GpuTimer rank_timer;

        for (int t = 0; t < k; ++t) {

            dev_Wt_vec_t = dev_W_ + t * R_C.rows; //u
            dev_Ht_vec_t = dev_H_ + t * R_C.cols; //v

            if (oiter > 1) {
                update_timer.Start();
                cudaStatus = updateRating(nThreadsPerBlock, nBlocks, R_C,
                                          Rt, dev_Rcol_ptr, dev_Rrow_idx, dev_Rcol_ptr_T, dev_Rrow_idx_T,
                                          dev_Rval, dev_Rval_t, dev_Wt_vec_t, dev_Ht_vec_t, true, cudaStatus);

                update_timer.Stop();
                update_time += update_timer.Elapsed();
            }

            rank_timer.Start();
            for (int iter = 1; iter <= parameters.maxinneriter; ++iter) {
                cudaStatus = RankOneUpdate(parameters, nThreadsPerBlock, nBlocks, lambda, R_C, Rt, dev_Rcol_ptr,
                                           dev_Rrow_idx, dev_Rcol_ptr_T, dev_Rrow_idx_T, dev_Rval,
                                           dev_Rval_t, dev_Wt_vec_t, dev_Ht_vec_t, cudaStatus);
            }
            rank_timer.Stop();
            rank_time += rank_timer.Elapsed();

            update_timer.Start();
            cudaStatus = updateRating(nThreadsPerBlock, nBlocks, R_C,
                                      Rt, dev_Rcol_ptr, dev_Rrow_idx, dev_Rcol_ptr_T, dev_Rrow_idx_T,
                                      dev_Rval, dev_Rval_t, dev_Wt_vec_t, dev_Ht_vec_t, false, cudaStatus);

            update_timer.Stop();
            update_time += update_timer.Elapsed();
        }

        update_time_acc += update_time;
        rank_time_acc += rank_time;
        /*********************Check RMSE*********************/
        rmse_timer.Start();

        gpuErrchk(hipMemset(d_rmse, 0, (T.nnz + 1) * sizeof(float)));
        gpuErrchk(hipMemset(d_pred_v, 0, (T.nnz + 1) * sizeof(float)));
        GPU_rmse<<<(T.nnz + 1023) / 1024, 1024>>>(d_test_row, d_test_col, d_test_val, d_pred_v, d_rmse,
                dev_W_, dev_H_, T.nnz, k, R_C.rows, R_C.cols, false);
        cudaStatus = hipGetLastError();
        gpuErrchk(cudaStatus);
        cudaStatus = hipDeviceSynchronize();
        gpuErrchk(cudaStatus);

        double tot_rmse = 0;
        double f_rmse = 0;
        gpuErrchk(hipMemcpy(rmse, d_rmse, (T.nnz + 1) * sizeof(float), hipMemcpyDeviceToHost));

        for (long i = 0; i < T.nnz; ++i) {
            tot_rmse += rmse[i];
        }
        f_rmse = sqrtf(tot_rmse / T.nnz);
        rmse_timer.Stop();

        float rmse_time = rmse_timer.Elapsed();
        printf("[-INFO-] iteration num %d \trank_time %.4lf|%.4lf s \tupdate_time %.4lf|%.4lfs \tRMSE=%lf time:%fs\n",
                oiter, rank_time, rank_time_acc, update_time, update_time_acc, f_rmse, rmse_time);
    }

    cudaStatus = hipMemcpy(H_, dev_H_, nbits_H_, hipMemcpyDeviceToHost);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(W_, dev_W_, nbits_W_, hipMemcpyDeviceToHost);
    gpuErrchk(cudaStatus);

    indexPosition = 0;
    for (long i = 0; i < k; ++i) {
        for (int j = 0; j < R_C.rows; ++j) {
            W[i][j] = W_[indexPosition];
            ++indexPosition;
        }
    }
    indexPosition = 0;
    for (long i = 0; i < k; ++i) {
        for (int j = 0; j < R_C.cols; ++j) {
            H[i][j] = H_[indexPosition];
            ++indexPosition;
        }
    }

    free(W_);
    free(H_);

    hipFree(dev_W_);
    hipFree(dev_H_);

    hipFree(dev_Rcol_ptr);
    hipFree(dev_Rrow_idx);
    hipFree(dev_Rcol_ptr_T);
    hipFree(dev_Rrow_idx_T);
    hipFree(dev_Rval);
    hipFree(dev_Rval_t);

    return cudaStatus;
}
