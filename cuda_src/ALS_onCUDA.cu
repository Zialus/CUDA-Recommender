#include "hip/hip_runtime.h"
#include "ALS_onCUDA.h"

// CUDA kernel to pause for at least num_cycle cycles
//__device__ void sleep(int64_t num_cycles) {
//    int64_t cycles = 0;
//    int64_t start = clock64();
//    while (cycles < num_cycles) {
//        cycles = clock64() - start;
//    }
//}

__device__ void choldc1_k(int n, float** a, float* p) {
    for (int i = 0; i < n; ++i) {
        for (int j = i; j < n; ++j) {
            float sum = a[i][j];
            for (int k = i - 1; k >= 0; --k) {
                sum -= a[i][k] * a[j][k];
            }
            if (i == j) {
                if (sum <= 0) {
                    printf(" a is not positive definite!\n");
                }
                p[i] = sqrtf(sum);
            } else {
                a[j][i] = sum / p[i];
            }
        }
    }
}

__device__ void choldcsl_k(int n, float** A) {
    float* p = (float*) malloc(n * sizeof(float));
    choldc1_k(n, A, p);
    for (int i = 0; i < n; ++i) {
        A[i][i] = 1 / p[i];
        for (int j = i + 1; j < n; ++j) {
            float sum = 0;
            for (int k = i; k < j; ++k) {
                sum -= A[j][k] * A[k][i];
            }
            A[j][i] = sum / p[j];
        }
    }
    free(p);
}

__device__ void inverseMatrix_CholeskyMethod_k(int n, float** A) {
    int i, j, k;
    choldcsl_k(n, A);
    for (i = 0; i < n; ++i) {
        for (j = i + 1; j < n; ++j) {
            A[i][j] = 0.0;
        }
    }
    for (i = 0; i < n; i++) {
        A[i][i] *= A[i][i];
        for (k = i + 1; k < n; ++k) {
            A[i][i] += A[k][i] * A[k][i];
        }
        for (j = i + 1; j < n; ++j) {
            for (k = j; k < n; ++k) {
                A[i][j] += A[k][i] * A[k][j];
            }
        }
    }
    for (i = 0; i < n; ++i) {
        for (j = 0; j < i; ++j) {
            A[i][j] = A[j][i];
        }
    }
}

//Multiply matrix M transpose by M 
__device__ void Mt_byM_multiply_k(long i, long j, float* H, float** Result, const long ptr, const long* idx) {
    float SUM;
    for (int I = 0; I < j; ++I) {
        for (int J = I; J < j; ++J) {
            SUM = 0.0f;
            for (int K = 0; K < i; ++K) {
                unsigned offset = idx[ptr + K] * j;
                //printf("%.3f %.3f\n", M[K][I], M[K][J]);
                //printf("%.3f %.3f\n", H[( offset) + I], H[( offset) + J]);
                SUM += H[offset + I] * H[offset + J];
            }
            Result[J][I] = SUM;
            Result[I][J] = SUM;
        }
    }
}

__global__ void updateW_overH_kernel(const long rows, const long* row_ptr, const long* col_idx, const long* colMajored_sparse_idx, const float* val, const float lambda, const unsigned k, float* W, float* H) {
    assert(row_ptr);
    assert(colMajored_sparse_idx);
    assert(val);
    assert(W);
    assert(H);

//    int tid = blockIdx.x * blockDim.x + threadIdx.x;

//    if (tid == 0) {
//        printf("OLA 1\n");
//    }

    //optimize W over H
    int ii = threadIdx.x + blockIdx.x * blockDim.x;
    for (int Rw = ii; Rw < rows; Rw += blockDim.x * gridDim.x) {
        //int offset_W = Rw*k;
        //int offset_H = Rw*cols;

        float* Wr = &W[Rw * k];
        unsigned omegaSize = row_ptr[Rw + 1] - row_ptr[Rw];
        float** subMatrix;
        float* subVector;

        if (omegaSize > 0) {
            subVector = (float*) malloc(k * sizeof(float));
            subMatrix = (float**) malloc(k * sizeof(float*));

//            if (tid == 0) {
//                printf("OLA 2\n");
//            }

            assert(subVector);
            assert(subMatrix);
            for (unsigned i = 0; i < k; ++i) {
//                if (tid == 0) {
//                    printf("OLA 3.1,i=%d\n", i);
//                } //else { sleep(1000000000);}
                subMatrix[i] = (float*) malloc(k * sizeof(float));
//                if (tid == 0) {
//                    printf("OLA 3.2,i=%d\n", i);
//                }
                assert(subMatrix);
            }

//            if (tid == 0) {
//                printf("OLA 4\n");
//            }

            Mt_byM_multiply_k(omegaSize, k, H, subMatrix, row_ptr[Rw], col_idx);

            //add lambda to diag of sub-matrix
            for (unsigned c = 0; c < k; c++) {
                subMatrix[c][c] = subMatrix[c][c] + lambda;
            }

            //invert sub-matrix
            inverseMatrix_CholeskyMethod_k(k, subMatrix);


            //sparse multiplication
            for (unsigned c = 0; c < k; ++c) {
                subVector[c] = 0;
                for (long idx = row_ptr[Rw]; idx < row_ptr[Rw + 1]; ++idx) {
                    unsigned idx2 = colMajored_sparse_idx[idx];
                    subVector[c] += val[idx2] * H[(col_idx[idx] * k) + c];
                }
            }

            //multiply subVector by subMatrix
            for (unsigned c = 0; c < k; ++c) {
                Wr[c] = 0;
                for (unsigned subVid = 0; subVid < k; ++subVid) {
                    Wr[c] += subVector[subVid] * subMatrix[c][subVid];
                }
            }


            for (unsigned i = 0; i < k; ++i) {
                free(subMatrix[i]);
            }
            free(subMatrix);
            free(subVector);
        } else {
            for (unsigned c = 0; c < k; ++c) {
                Wr[c] = 0.0f;
            }
        }
    }
}

__global__ void updateH_overW_kernel(const long cols, const long* col_ptr, const long* row_idx, const float* val, const float lambda, const unsigned k, float* W, float* H) {
    //optimize H over W
    int ii = threadIdx.x + blockIdx.x * blockDim.x;
    for (int Rh = ii; Rh < cols; Rh += blockDim.x * gridDim.x) {
        float* Hr = &H[Rh * k];
        //int offset_H = Rh*k;
        unsigned omegaSize = col_ptr[Rh + 1] - col_ptr[Rh];
        float** subMatrix;// ** W_Omega;
        float* subVector;

        if (omegaSize > 0) {
            subVector = (float*) malloc(k * sizeof(float));
            subMatrix = (float**) malloc(k * sizeof(float*));
            for (unsigned i = 0; i < k; ++i) {
                subMatrix[i] = (float*) malloc(k * sizeof(float));
            }

            Mt_byM_multiply_k(omegaSize, k, W, subMatrix, col_ptr[Rh], row_idx);

            //add lambda to diag of sub-matrix
            for (unsigned c = 0; c < k; c++) {
                subMatrix[c][c] = subMatrix[c][c] + lambda;
            }

            //invert sub-matrix
            inverseMatrix_CholeskyMethod_k(k, subMatrix);


            //sparse multiplication
            for (unsigned c = 0; c < k; ++c) {
                subVector[c] = 0;
                for (long idx = col_ptr[Rh]; idx < col_ptr[Rh + 1]; ++idx) {
                    subVector[c] += val[idx] * W[(row_idx[idx] * k) + c];
                }
            }

            //multiply subVector by subMatrix
            for (unsigned c = 0; c < k; ++c) {
                Hr[c] = 0;
                for (unsigned subVid = 0; subVid < k; ++subVid) {
                    Hr[c] += subVector[subVid] * subMatrix[c][subVid];
                }
            }


            for (unsigned i = 0; i < k; ++i) {
                free(subMatrix[i]);
            }
            free(subMatrix);
            free(subVector);
        } else {
            for (unsigned c = 0; c < k; ++c) {
                Hr[c] = 0.0f;
            }
        }
    }
}

void kernel_wrapper_als_NV(smat_t& R, testset_t& T, mat_t& W, mat_t& H, parameter& parameters) {
    hipError_t cudaStatus;
    // Reset GPU.
    cudaStatus = hipDeviceReset();
    gpuErrchk(cudaStatus);
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    gpuErrchk(cudaStatus);

    cudaStatus = als_NV(R, T, W, H, parameters);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ALS FAILED: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipDeviceReset();
    gpuErrchk(cudaStatus);
}

hipError_t als_NV(smat_t& R_C, testset_t& T, mat_t& W, mat_t& H, parameter& parameters) {
    int nThreadsPerBlock = parameters.nThreadsPerBlock;
    int nBlocks = parameters.nBlocks;

    hipError_t cudaStatus;

    float lambda = parameters.lambda;
    int k = parameters.k;

    long* dev_col_ptr = nullptr;
    long* dev_row_ptr = nullptr;
    long* dev_row_idx = nullptr;
    long* dev_col_idx = nullptr;
    long* dev_colMajored_sparse_idx = nullptr;
    float* dev_val = nullptr;

    float* dev_W_ = nullptr;
    float* dev_H_ = nullptr;


    size_t nbits_W_ = R_C.rows * k * sizeof(float);
    float* W_ = (float*) malloc(nbits_W_);
    size_t nbits_H_ = R_C.cols * k * sizeof(float);
    float* H_ = (float*) malloc(nbits_H_);

    int indexPosition = 0;
    for (long i = 0; i < R_C.rows; ++i) {
        for (int j = 0; j < k; ++j) {
            W_[indexPosition] = W[i][j];
            ++indexPosition;
        }
    }

    indexPosition = 0;
    for (long i = 0; i < R_C.cols; ++i) {
        for (int j = 0; j < k; ++j) {
            H_[indexPosition] = H[i][j];
            ++indexPosition;
        }
    }

    cudaStatus = hipMalloc((void**) &dev_W_, nbits_W_);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_H_, nbits_H_);
    gpuErrchk(cudaStatus);

    cudaStatus = hipMemcpy(dev_W_, W_, nbits_W_, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_H_, H_, nbits_H_, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);


    cudaStatus = hipMalloc((void**) &dev_col_ptr, R_C.nbits_col_ptr);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_row_ptr, R_C.nbits_row_ptr);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_row_idx, R_C.nbits_row_idx);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_col_idx, R_C.nbits_col_idx);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_colMajored_sparse_idx, R_C.nbits_colMajored_sparse_idx);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMalloc((void**) &dev_val, R_C.nbits_val);
    gpuErrchk(cudaStatus);


    cudaStatus = hipMemcpy(dev_col_ptr, R_C.col_ptr, R_C.nbits_col_ptr, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_row_ptr, R_C.row_ptr, R_C.nbits_row_ptr, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_row_idx, R_C.row_idx, R_C.nbits_row_idx, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_col_idx, R_C.col_idx, R_C.nbits_col_idx, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_colMajored_sparse_idx, R_C.colMajored_sparse_idx, R_C.nbits_colMajored_sparse_idx, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(dev_val, R_C.val, R_C.nbits_val, hipMemcpyHostToDevice);
    gpuErrchk(cudaStatus);


    float* rmse = (float*) malloc((T.nnz) * sizeof(float));

    long* d_test_row;
    long* d_test_col;
    float* d_test_val;
    float* d_pred_v;
    float* d_rmse;

    gpuErrchk(hipMalloc((void**) &d_test_row, (T.nnz + 1) * sizeof(long)));
    gpuErrchk(hipMalloc((void**) &d_test_col, (T.nnz + 1) * sizeof(long)));
    gpuErrchk(hipMalloc((void**) &d_test_val, (T.nnz + 1) * sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_pred_v, (T.nnz + 1) * sizeof(float)));
    gpuErrchk(hipMalloc((void**) &d_rmse, (T.nnz + 1) * sizeof(float)));

    gpuErrchk(hipMemcpy(d_test_row, T.test_row, (T.nnz + 1) * sizeof(long), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_test_col, T.test_col, (T.nnz + 1) * sizeof(long), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_test_val, T.test_val, (T.nnz + 1) * sizeof(float), hipMemcpyHostToDevice));

    for (int iter = 1; iter <= parameters.maxiter; ++iter) {

        GpuTimer t;
        t.Start();
        /********************optimize W over H***************/
        updateW_overH_kernel<<<nBlocks, nThreadsPerBlock>>>(R_C.rows, dev_row_ptr, dev_col_idx,
                dev_colMajored_sparse_idx, dev_val, lambda, k, dev_W_, dev_H_);
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        gpuErrchk(cudaStatus);
        cudaStatus = hipDeviceSynchronize();
        gpuErrchk(cudaStatus);

        /*******************optimize H over W****************/
        updateH_overW_kernel<<<nBlocks, nThreadsPerBlock>>>(R_C.cols, dev_col_ptr, dev_row_idx,
                dev_val, lambda, k, dev_W_, dev_H_);
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        gpuErrchk(cudaStatus);
        cudaStatus = hipDeviceSynchronize();
        gpuErrchk(cudaStatus);
        t.Stop();
        /*********************Check RMSE*********************/
        gpuErrchk(hipMemset(d_rmse, 0, (T.nnz + 1) * sizeof(float)));
        gpuErrchk(hipMemset(d_pred_v, 0, (T.nnz + 1) * sizeof(float)));
        GPU_rmse<<<(T.nnz + 1023) / 1024, 1024>>>(d_test_row, d_test_col, d_test_val, d_pred_v, d_rmse,
                dev_W_, dev_H_, T.nnz, k, R_C.rows, R_C.cols, true);
        cudaStatus = hipGetLastError();
        gpuErrchk(cudaStatus);
        cudaStatus = hipDeviceSynchronize();
        gpuErrchk(cudaStatus);

        float tot_rmse = 0;
        float f_rmse = 0;
        gpuErrchk(hipMemcpy(rmse, d_rmse, (T.nnz + 1) * sizeof(float), hipMemcpyDeviceToHost));

        for (unsigned i = 0; i < T.nnz; ++i) {
            tot_rmse += rmse[i];
        }
        f_rmse = sqrtf(tot_rmse / T.nnz);
        printf("iter %d RMSE %f time %f\n", iter, f_rmse, t.Elapsed());
    }

    cudaStatus = hipMemcpy(H_, dev_H_, nbits_H_, hipMemcpyDeviceToHost);
    gpuErrchk(cudaStatus);
    cudaStatus = hipMemcpy(W_, dev_W_, nbits_W_, hipMemcpyDeviceToHost);
    gpuErrchk(cudaStatus);

    indexPosition = 0;
    for (long i = 0; i < R_C.rows; ++i) {
        for (int j = 0; j < k; ++j) {
            W[i][j] = W_[indexPosition];
            ++indexPosition;
        }
    }
    indexPosition = 0;
    for (long i = 0; i < R_C.cols; ++i) {
        for (int j = 0; j < k; ++j) {
            H[i][j] = H_[indexPosition];
            ++indexPosition;
        }
    }

    free(W_);
    free(H_);

    hipFree(dev_W_);
    hipFree(dev_H_);

    hipFree(dev_col_ptr);
    hipFree(dev_row_ptr);
    hipFree(dev_row_idx);
    hipFree(dev_col_idx);
    hipFree(dev_colMajored_sparse_idx);
    hipFree(dev_val);

    return cudaStatus;
}
